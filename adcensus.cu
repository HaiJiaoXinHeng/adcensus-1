#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>

#define TB 128

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

#define COLOR_DIFF(x, i, j) \
	max(abs(x[(i)]               - x[(j)]), \
    max(abs(x[(i) +   dim2*dim3] - x[(j) +   dim2*dim3]), \
	    abs(x[(i) + 2*dim2*dim3] - x[(j) + 2*dim2*dim3])))

__global__ void ad(float *x0, float *x1, float *output, int size, int size3, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		int xy = d % size23;
		d /= size23;

		float dist = 0;
		if (x - d < 0) {
			dist = HIP_INF;
		} else {
			for (int i = 0; i < 3; i++) {
				int ind = i * size23 + xy;
				dist += fabsf(x0[ind] - x1[ind - d]);
			}
		}
		output[id] = dist / 3;
	}
}

int ad(lua_State *L)
{
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");

	ad<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(x0),
		THCudaTensor_data(x1),
		THCudaTensor_data(output),
		THCudaTensor_nElement(output),
		THCudaTensor_size(output, 3),
		THCudaTensor_size(output, 2) * THCudaTensor_size(output, 3));
	checkCudaError(L);
	return 0;
}


__global__ void census(float *x0, float *x1, float *output, int size, int size2, int size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;

		float dist;
		if (x - d < 0) {
			dist = HIP_INF;
		} else {
			dist = 0;
			for (int i = 0; i < 3; i++) {
				int ind_p = (i * size2 + y) * size3 + x;
				for (int yy = y - 3; yy <= y + 3; yy++) {
					for (int xx = x - 4; xx <= x + 4; xx++) {
						if (0 <= xx - d && xx < size3 && 0 <= yy && yy < size2) {
							int ind_q = (i * size2 + yy) * size3 + xx;
							if ((x0[ind_q] < x0[ind_p]) != (x1[ind_q - d] < x1[ind_p - d])) {
								dist++;
							}
						} else {
							dist++;
						}
					}
				}
			}
		}
		output[id] = dist / 3;
	}
}


int census(lua_State *L)
{
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");

	census<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(x0),
		THCudaTensor_data(x1),
		THCudaTensor_data(output),
		THCudaTensor_nElement(output),
		THCudaTensor_size(output, 2),
		THCudaTensor_size(output, 3));
	checkCudaError(L);
	return 0;
}

__global__ void spatial_argmin(float *input, float *output, int size, int size1, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		int argmin = 0;
		float min = 2e38;
		for (int i = 0; i < size1; i++) {
			float val = input[(dim0 * size1 + i) * size23 + dim23];
			if (val < min) {
				min = val;
				argmin = i;
			}
		}
		output[id] = argmin + 1;
	}
}

int spatial_argmin(lua_State *L)
{
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	spatial_argmin<<<(THCudaTensor_nElement(output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(input),
		THCudaTensor_data(output),
		THCudaTensor_nElement(output),
		THCudaTensor_size(input, 1),
		THCudaTensor_size(input, 2) * THCudaTensor_size(output, 3));
	checkCudaError(L);
	return 0;
}

/* median 3x3 filter */
__global__ void median3(float *img, float *out, int size, int height, int width)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		const int x = id % width;
		const int y = id / width;

		float w[9] = {
			y == 0          || x == 0         ? 0 : img[id - width - 1],
			y == 0                            ? 0 : img[id - width],
			y == 0          || x == width - 1 ? 0 : img[id - width + 1],
			                   x == 0         ? 0 : img[id         - 1],
			                                        img[id],
			                   x == width - 1 ? 0 : img[id         + 1],
			y == height - 1 || x == 0         ? 0 : img[id + width - 1],
			y == height - 1                   ? 0 : img[id + width],
			y == height - 1 || x == width - 1 ? 0 : img[id + width + 1]
		};

		for (int i = 0; i < 5; i++) {
			float tmp = w[i];
			int idx = i;
			for (int j = i + 1; j < 9; j++) {
				if (w[j] < tmp) {
					idx = j;
					tmp = w[j];
				}
			}
			w[idx] = w[i];
			w[i] = tmp;
		}

		out[id] = w[4];
	}
}

int median3(lua_State *L)
{
	THCudaTensor *img = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	median3<<<(THCudaTensor_nElement(img) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(img),
		THCudaTensor_data(out),
		THCudaTensor_nElement(img),
		THCudaTensor_size(img, 2),
		THCudaTensor_size(img, 3));
	checkCudaError(L);
	return 0;
}

__global__ void cross(float *x0, float *vol, float *out, int size, int dim2, int dim3, int L1, int L2, float tau1, float tau2)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dir = id;
		int x = dir % dim3;
		dir /= dim3;
		int y = dir % dim2;
		dir /= dim2;

		int dx = 0;
		int dy = 0;
		if (dir == 0) {
			dx = -1;
		} else if (dir == 1) {
			dx = 1;
		} else if (dir == 2) {
			dy = -1;
		} else if (dir == 3) {
			dy = 1;
		} else {
			assert(0);
		}

		int xx, yy, ind1, ind2, ind3, dist;
		ind1 = y * dim3 + x;
		for (xx = x + dx, yy = y + dy;;xx += dx, yy += dy) {
			if (xx < 0 || xx >= dim3 || yy < 0 || yy >= dim2) break;

			dist = max(abs(xx - x), abs(yy - y));
			if (dist == 1) continue;

			ind2 = yy * dim3 + xx;
			ind3 = (yy - dy) * dim3 + (xx - dx);

			/* rule 1 */
			if (COLOR_DIFF(x0, ind1, ind2) >= tau1) break;
			if (COLOR_DIFF(x0, ind2, ind3) >= tau1) break;

			/* rule 2 */
			if (dist >= L1) break;

			/* rule 3 */
			if (dist >= L2) {
				if (COLOR_DIFF(x0, ind1, ind2) >= tau2) break;
			}
		}
		out[id] = dir <= 1 ? xx : yy;
	}
}

int cross(lua_State *L)
{
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *vol = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int L1 = luaL_checkinteger(L, 4);
	int L2 = luaL_checkinteger(L, 5);
	float tau1 = luaL_checknumber(L, 6);
	float tau2 = luaL_checknumber(L, 7);

	cross<<<(THCudaTensor_nElement(out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(x0),
		THCudaTensor_data(vol),
		THCudaTensor_data(out),
		THCudaTensor_nElement(out),
		THCudaTensor_size(out, 2),
		THCudaTensor_size(out, 3),
		L1, L2, tau1, tau2);

	checkCudaError(L);
	return 0;
}


__global__ void cbca(float *x0c, float *x1c, float *vol, float *out, int size, int dim2, int dim3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = id;
		int x = d % dim3;
		d /= dim3;
		int y = d % dim2;
		d /= dim2;

		if (x - d < 0) {
			out[id] = vol[id];
		} else {
			float sum = 0;
			int cnt = 0;

			if (direction == 0) {
				int xx_s = max(x0c[(0 * dim2 + y) * dim3 + x], x1c[(0 * dim2 + y) * dim3 + x - d] + d);
				int xx_t = min(x0c[(1 * dim2 + y) * dim3 + x], x1c[(1 * dim2 + y) * dim3 + x - d] + d);
				for (int xx = xx_s + 1; xx < xx_t; xx++) {
					int yy_s = max(x0c[(2 * dim2 + y) * dim3 + xx], x1c[(2 * dim2 + y) * dim3 + xx - d]);
					int yy_t = min(x0c[(3 * dim2 + y) * dim3 + xx], x1c[(3 * dim2 + y) * dim3 + xx - d]);
					for (int yy = yy_s + 1; yy < yy_t; yy++) {
						sum += vol[(d * dim2 + yy) * dim3 + xx];
						cnt++;
					}
				}
			} else {
				int yy_s = max(x0c[(2 * dim2 + y) * dim3 + x], x1c[(2 * dim2 + y) * dim3 + x - d]);
				int yy_t = min(x0c[(3 * dim2 + y) * dim3 + x], x1c[(3 * dim2 + y) * dim3 + x - d]);
				for (int yy = yy_s + 1; yy < yy_t; yy++) {
					int xx_s = max(x0c[(0 * dim2 + yy) * dim3 + x], x1c[(0 * dim2 + yy) * dim3 + x - d] + d);
					int xx_t = min(x0c[(1 * dim2 + yy) * dim3 + x], x1c[(1 * dim2 + yy) * dim3 + x - d] + d);
					for (int xx = xx_s + 1; xx < xx_t; xx++) {
						sum += vol[(d * dim2 + yy) * dim3 + xx];
						cnt++;
					}
				}
			}

			assert(cnt > 0);
			out[id] = sum / cnt;
		}
	}
}


int cbca(lua_State *L)
{
	THCudaTensor *x0c = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1c = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *vol1 = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *vol2 = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");

	for (int i = 0; i < 4; i++) {
		cbca<<<(THCudaTensor_nElement(vol2) - 1) / TB + 1, TB>>>(
			THCudaTensor_data(x0c),
			THCudaTensor_data(x1c),
			THCudaTensor_data(i % 2 == 0 ? vol1 : vol2),
			THCudaTensor_data(i % 2 == 0 ? vol2 : vol1),
			THCudaTensor_nElement(vol2),
			THCudaTensor_size(vol2, 2),
			THCudaTensor_size(vol2, 3),
			i % 2);
	}
	checkCudaError(L);
	return 0;
}

__global__ void sgm(float *x0, float *x1, float *vol, float *out, int dim1, int dim2, int dim3, float pi1, float pi2, float tau_so, int direction)
{
	int x, y, dx, dy;

	dx = dy = 0;
	if (direction <= 1) {
		y = blockIdx.x * blockDim.x + threadIdx.x;
		if (y >= dim2) {
			return;
		}
		if (direction == 0) {
			x = 0;
			dx = 1;
		} else if (direction == 1) {
			x = dim3 - 1;
			dx = -1;
		}
	} else {
		x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x >= dim3) {
			return;
		}
		if (direction == 2) {
			y = 0;
			dy = 1;
		} else if (direction == 3) {
			y = dim2 - 1;
			dy = -1;
		}
	}

	float min_prev = HIP_INF;
	for (; 0 <= y && y < dim2 && 0 <= x && x < dim3; x += dx, y += dy) {
		float min_curr = HIP_INF;
		for (int d = 0; d < dim1; d++) {
			int ind = (d * dim2 + y) * dim3 + x;
			if (x - d < 0 || y - dy < 0 || y - dy >= dim2 || x - d - dx < 0 || x - dx >= dim3) {
				out[ind] = vol[ind];
			} else {
				int ind2 = y * dim3 + x;

				float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * dim3 - dx);
				float D2 = COLOR_DIFF(x1, ind2 - d, ind2 - d - dy * dim3 - dx);
				float P1, P2;
				if (D1 < tau_so && D2 < tau_so) { 
					P1 = pi1; 
					P2 = pi2; 
				} else if (D1 > tau_so && D2 > tau_so) { 
					P1 = pi1 / 10; 
					P2 = pi2 / 10; 
				} else {
					P1 = pi1 / 4;
					P2 = pi2 / 4;
				}

				assert(min_prev != HIP_INF);
				float cost = min(out[ind - dy * dim3 - dx], min_prev + P2);
				if (d > 0) {
					cost = min(cost, out[ind - dim2 * dim3 - dy * dim3 - dx] + P1);
				}
				if (d < dim1 - 1) {
					cost = min(cost, out[ind + dim2 * dim3 - dy * dim3 - dx] + P1);
				}
				out[ind] = vol[ind] + cost - min_prev;
			}
			if (out[ind] < min_curr) {
				min_curr = out[ind];
			}
		}
		min_prev = min_curr;
	}
}

int sgm(lua_State *L)
{
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *vol = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	float pi1 = luaL_checknumber(L, 5);
	float pi2 = luaL_checknumber(L, 6);
	float tau_so = luaL_checknumber(L, 7);

	int dim1 = THCudaTensor_size(out, 1);
	int dim2 = THCudaTensor_size(out, 2);
	int dim3 = THCudaTensor_size(out, 3);


	for (int i = 0; i < 4; i++) {
		hipStream_t stream;
		hipStreamCreate(&stream);
		sgm<<<(THCudaTensor_size(vol, 2) - 1) / TB + 1, TB, 0, stream>>>(
			THCudaTensor_data(x0),
			THCudaTensor_data(x1),
			THCudaTensor_data(vol),
			THCudaTensor_data(out) + i * dim1 * dim2 * dim3,
			dim1, dim2, dim3, pi1, pi2, tau_so, i);
		hipStreamDestroy(stream);
	}

	checkCudaError(L);
	return 0;
}

static const struct luaL_Reg funcs[] = {
	{"ad", ad},
	{"cbca", cbca},
	{"census", census},
	{"cross", cross},
	{"median3", median3},
	{"spatial_argmin", spatial_argmin},
	{"sgm", sgm},
	{NULL, NULL}
};

extern "C" int luaopen_libadcensus(lua_State *L) {
	luaL_openlib(L, "adcensus", funcs, 0);
	return 1;
}
